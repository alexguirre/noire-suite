#include "hip/hip_runtime.h"
#include "crc.cuh"
#include <cstddef>
#include <cstdint>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <math.h>
#include <random>

template<std::size_t N>
struct fixedString
{
	char buffer[N];

	__host__ __device__ std::size_t capacity() const { return N; }
	__host__ __device__ std::size_t size() const
	{
		std::size_t i = 0;
		while (i < N && buffer[i])
		{
			i++;
		}

		return i;
	}
};

template<std::size_t N, std::size_t StrSize>
struct fixedStringArray
{
	fixedString<StrSize> strings[N];

	__host__ __device__ std::size_t capacity() const { return N; }
	__host__ __device__ std::size_t size() const { return N; }
};

using stringArray = fixedStringArray<8192*128, 64>;

// function to add the elements of two
__global__ void calcHashes(stringArray* strings, std::uint32_t* hashes)
{
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	const int stride = blockDim.x * gridDim.x;

	for (int i = index; i < strings->size(); i += stride)
	{
		hashes[i] = crc32(strings->strings[i].buffer, strings->strings[i].size());
	}
}

int main(void)
{
	stringArray* strings;
	hipMallocManaged(&strings, sizeof(*strings));
	hipMemset(strings, 0, sizeof(*strings));

	std::uint32_t* hashes;
	hipMallocManaged(&hashes, strings->size() * sizeof(*hashes));
	hipMemset(hashes, 0, strings->size() * sizeof(*hashes));

	std::random_device rndDev{};
	std::mt19937 rndEng{ rndDev() };
	std::uniform_int_distribution<std::mt19937::result_type> rndDist(48, 122);

	for (std::size_t i = 0; i < strings->size(); i++)
	{
		auto& s = strings->strings[i];
		for (std::size_t j = 0; j < s.capacity() - 1; j++)
		{
			s.buffer[j] = rndDist(rndEng);
		}
	}

	// Run kernel on 1M elements on the CPU
	int BlockSize = 256;
	int NumBlocks = (strings->size() + BlockSize - 1) / BlockSize;
	calcHashes<<<NumBlocks, BlockSize>>>(strings, hashes);

	hipDeviceSynchronize();

	/*for (std::size_t i = 0; i < strings->size(); i++)
	{
		auto& s = strings->strings[i];
		printf("#%d: %s = 0x%08X\n", i, s.buffer, hashes[i]);
	}*/

	// Free memory
	hipFree(strings);
	hipFree(hashes);

	return 0;
}